#include "hip/hip_runtime.h"


__global__ void add1(float* arr)
{
	int t_id = threadIdx.x;
	arr[t_id] = arr[t_id] * arr[t_id];
}

void square(float* arr, int size)
{
	float* dev_arr;
	hipMalloc(&dev_arr, size * sizeof(float));
	hipMemcpy(dev_arr, arr, size * sizeof(float), hipMemcpyHostToDevice);
	add1 <<<1, size >>> (dev_arr);
	hipMemcpy(arr, dev_arr, size * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(dev_arr);
}