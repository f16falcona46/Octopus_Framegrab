#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hipfft/hipfft.h"
#include "CUDAStreamer.h"

#ifdef NDEBUG
#define gpuErrchk(ans) ans
#else
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		//if (abort) exit(code);
	}
}
#endif

__global__ void Interpolate(const CUDAStreamer::Consumer_element_t* in, float* out, const int* indexes, const float* fractions, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) out[idx] = in[indexes[idx]] * (1.0f - fractions[idx]) + in[indexes[idx] + 1] * fractions[idx];
}

__global__ void NormAndCopy(const hipfftComplex* in, CUDAStreamer::Producer_element_t* out, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)	out[idx] = sqrt(in[idx].x * in[idx].x + in[idx].y * in[idx].y);
}

void CUDAStreamer::DoFFT(CUDAStreamer* streamer)
{
	Interpolate<<<(streamer->m_bufcount + 32 - 1) / 32, 32 >>>(streamer->m_device_in_buf, streamer->m_device_conv_in_buf, streamer->m_device_lerp_index, streamer->m_device_lerp_fraction, streamer->m_bufcount);
	if (hipfftExecR2C(streamer->m_plan, streamer->m_device_conv_in_buf, streamer->m_device_out_buf) != HIPFFT_SUCCESS) throw std::runtime_error("Failed to perform FFT.");
	NormAndCopy <<<(streamer->m_bufcount + 32 - 1) / 32, 32 >>> (streamer->m_device_out_buf, streamer->m_device_norm_out_buf, streamer->m_bufcount);
}
